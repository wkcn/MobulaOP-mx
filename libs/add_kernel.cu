#include <iostream>
#include "gpu_add.hpp"

MOBULA_KERNEL add_kernel(const int n, const float *a, const float *b, float *output){
	KERNEL_LOOP(i, n){
		output[i] = a[i] + b[i];
	}
}

void set_device(int device_id) {
    int current_device;
    CUDA_CHECK(hipGetDevice(&current_device));
    if (current_device != device_id) {
        CUDA_CHECK(hipSetDevice(device_id));
    }
}

void gpu_add(const float *a, const float *b, int n, float *c, int device_id) {
    set_device(device_id);
    KERNEL_RUN(add_kernel, n)(n, a, b, c);
}
